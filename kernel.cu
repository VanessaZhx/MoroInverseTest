#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <chrono>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    std::cout << "\nError at "<<__FILE__<<":"<<__LINE__<<": "<<x<<"\n"; \
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    std::cout << "\nError at "<<__FILE__<<":"<<__LINE__<<": "<<x<<"\n"; \
    return EXIT_FAILURE;}} while(0)

#define EXP_TIMES 100

using namespace std;

__global__ void moro_inv(float* data, int cnt, float mean, float std) {
    // Each thread will handle one transfer
    size_t Idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (Idx >= cnt) return;

    data[Idx] = normcdfinvf(data[Idx]) * std + mean;
}

int main()
{
    int m = 1 << 12;
    int n = 1 << 12;    // Test with 16M data, generate (4k * 4k) sobol sequence

    //allocate host memory
    /*size_t bytes = size * sizeof(int);
    int* idata_host = (int*)malloc(bytes);
    int* odata_host = (int*)malloc(grid.x * sizeof(int));
    int* tmp = (int*)malloc(bytes);*/

    // Allocate memory
    size_t bytes = m * n * sizeof(float);
    // host
    float *sepr_host = (float*)malloc(bytes);    // Seperate sobol and moro
    float *comb_host = (float*)malloc(bytes);   // Combine sobol and moro
    // device
    float *sepr_dev = NULL;
    float *comb_dev = NULL;
    CUDA_CALL(hipMalloc((void**)&sepr_dev, bytes));
    CUDA_CALL(hipMalloc((void**)&comb_dev, bytes));

    // Generator
    
    int offset = 1024;

    // Seperate 
    // Set up block manually
    int blocksize = 1024;
    dim3 block(blocksize, 1);
    dim3 grid((m * n - 1) / block.x + 1, 1);
    printf("grid %d block %d \n", grid.x, block.x);

    chrono::steady_clock::time_point start, end;
    start = chrono::steady_clock::now();
    hiprandGenerator_t gen_sepr;
    CURAND_CALL(hiprandCreateGenerator(&gen_sepr, HIPRAND_RNG_QUASI_SOBOL32));
    CURAND_CALL(hiprandSetGeneratorOffset(gen_sepr, offset));
    CURAND_CALL(hiprandSetQuasiRandomGeneratorDimensions(gen_sepr, m));
    CURAND_CALL(hiprandGenerateUniform(gen_sepr, sepr_dev, n * m));
    moro_inv << < grid, block >> > (sepr_dev, m * n, 0, 0.5);
    hipDeviceSynchronize();
    end = chrono::steady_clock::now();
    chrono::duration<double, std::milli> elapsed = end - start;

    cout << "Seperated version EXE TIME: " << elapsed.count() << "ms" << endl;

    CUDA_CALL(hipMemcpy(sepr_host, sepr_dev, bytes, hipMemcpyDeviceToHost));
    CUDA_CALL(hiprandDestroyGenerator(gen_sepr));


    // Combined
    start = chrono::steady_clock::now();
    hiprandGenerator_t gen_comb;
    CURAND_CALL(hiprandCreateGenerator(&gen_comb, HIPRAND_RNG_QUASI_SOBOL32));
    CURAND_CALL(hiprandSetGeneratorOffset(gen_comb, offset));
    CURAND_CALL(hiprandSetQuasiRandomGeneratorDimensions(gen_comb, m));
    CURAND_CALL(hiprandGenerateNormal(gen_comb, comb_dev, n * m, 0, 0.5));
    end = chrono::steady_clock::now();
    elapsed = end - start;

    cout << "Combined version EXE TIME: " << elapsed.count() << "ms" << endl;


    CUDA_CALL(hipMemcpy(comb_host, comb_dev, bytes, hipMemcpyDeviceToHost));
    CUDA_CALL(hiprandDestroyGenerator(gen_comb));

    // Correction check
    bool pass = true;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (abs(comb_host[i * n + j] - sepr_host[i * n + j]) > 1e-6) {
                pass = false;
                cout << i << " "<< j << endl;
                cout << comb_host[i * n + j] << " "<< sepr_host[i * n + j] << endl;
                break;
            }
        }
        if (!pass) break;
    }
    if (!pass) {
        cout << "CORRECTION CHECK: FAILED" << endl;
    }
    else {
        cout << "CORRECTION CHECK: PASS" << endl;

        // loop to get average time
        double sepr_time = 0;
        double comb_time = 0;

        for (int i = 0; i < EXP_TIMES; i++) {
            
            // Seperated
            // ------------------------
            start = chrono::steady_clock::now();

            CURAND_CALL(hiprandCreateGenerator(&gen_sepr, HIPRAND_RNG_QUASI_SOBOL32));
            CURAND_CALL(hiprandSetGeneratorOffset(gen_sepr, offset));
            CURAND_CALL(hiprandSetQuasiRandomGeneratorDimensions(gen_sepr, m));
            CURAND_CALL(hiprandGenerateUniform(gen_sepr, sepr_dev, n * m));
            moro_inv << < grid, block >> > (sepr_dev, m * n, 0, 0.5);
            hipDeviceSynchronize();
            CUDA_CALL(hiprandDestroyGenerator(gen_sepr));

            end = chrono::steady_clock::now();
            elapsed = end - start;
            sepr_time += elapsed.count();
            // ------------------------
            

            // Combined
            // ------------------------
            start = chrono::steady_clock::now();
            CURAND_CALL(hiprandCreateGenerator(&gen_comb, HIPRAND_RNG_QUASI_SOBOL32));
            CURAND_CALL(hiprandSetGeneratorOffset(gen_comb, offset));
            CURAND_CALL(hiprandSetQuasiRandomGeneratorDimensions(gen_comb, m));
            CURAND_CALL(hiprandGenerateNormal(gen_comb, comb_dev, n * m, 0, 0.5));
            CUDA_CALL(hiprandDestroyGenerator(gen_comb));
            end = chrono::steady_clock::now();
            elapsed = end - start;
            comb_time += elapsed.count();
            // ------------------------
        }

        cout << "Seperated version average EXE TIME: " << sepr_time / EXP_TIMES << "ms" << endl;;
        cout << "Combined version average EXE TIME: " << comb_time / EXP_TIMES << "ms" << endl;;
    }  

    free(comb_host);
    free(sepr_host);
    CUDA_CALL(hipFree(sepr_dev));
    CUDA_CALL(hipFree(comb_dev));

    return 0;
}